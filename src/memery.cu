#include "cuda_tools.h"

int main() {
    int device = 0;
    checkCudaRuntime(hipSetDevice(device));

    // global memory
    float* memory_device = nullptr;
    checkCudaRuntime(hipMalloc(&memory_device, 100*sizeof(float)));

    // pageable memory
    float* memory_host = new float[100];
    memory_host[2] = 520.25;
    checkCudaRuntime(hipMemcpy(memory_device, memory_host, 100*sizeof(float), hipMemcpyHostToDevice));

    // pinned memory
    // Note: dst, src
    float* memory_page_locked = nullptr;
    checkCudaRuntime(hipHostMalloc(&memory_page_locked, 100 * sizeof(float)));
    checkCudaRuntime(hipMemcpy(memory_page_locked, memory_device, 100*sizeof(float), hipMemcpyDeviceToHost));

    printf("%f\n", memory_page_locked[2]);
    delete[] memory_host;
    checkCudaRuntime(hipFree(memory_device));
    return 0;
}