#include "cuda_tools.h"

int main() {
    int device_id = 0;
    checkCudaRuntime(hipSetDevice(device_id));

    hipStream_t stream = nullptr;
    checkCudaRuntime(hipStreamCreate(&stream));

    // 在GPU上开辟一个 100 * float 的空间
    size_t size = 100 * sizeof(float);
    float* device_mem = nullptr;
    checkCudaRuntime(hipMalloc(&device_mem, size));

    // 在CPU开辟空间，并将数据存入到GPU
    float* host_mem = nullptr;
    checkCudaRuntime(hipHostMalloc(&host_mem, size));
    host_mem[2] = 520.25;
    // 通过流异步操作
    checkCudaRuntime(hipMemcpyAsync(device_mem, host_mem, size, hipMemcpyHostToDevice, stream));

    // 在CPU上开辟空间，将数据从GPU拷贝到CPU
    float* pin_mem_host = nullptr;
    checkCudaRuntime(hipHostMalloc(&pin_mem_host, size));
    checkCudaRuntime(hipMemcpyAsync(pin_mem_host, device_mem, size, hipMemcpyDeviceToHost));

    // 等待流中的所有任务进行完
    checkCudaRuntime(hipStreamSynchronize(stream));

    printf("%f\n", pin_mem_host[2]);

    // 释放
    checkCudaRuntime(hipFree(device_mem));
    checkCudaRuntime(hipHostFree(pin_mem_host));
    checkCudaRuntime(hipHostFree(host_mem));

    // 释放流
    checkCudaRuntime(hipStreamDestroy(stream));

    return 0;
}